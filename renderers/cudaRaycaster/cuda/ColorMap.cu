#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2016  Ahmet Bilgili <ahmetbilgili@gmail.com>
 *
 * This file is part of Livre <https://github.com/bilgili/Libre>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "ColorMap.cuh"
#include "debug.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_texture_types.h>

namespace livre
{
namespace cuda
{
ColorMap::ColorMap()
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc< float4 >();
    checkCudaErrors( hipMallocArray( &_array, &channelDesc, 256, 1));

    upload( lexis::render::ColorMap::getDefaultColorMap( 0.0f, 256.0f ));

    // create texture object
    hipResourceDesc resDesc;
    ::memset( &resDesc, 0, sizeof( hipResourceDesc ));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = _array;

    hipTextureDesc texDesc;
    ::memset( &texDesc, 0, sizeof( hipTextureDesc ));
    texDesc.readMode = hipReadModeElementType;
    texDesc.addressMode[ 0 ] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.normalizedCoords = 1;

    // create texture object: we only have to do this once!
    checkCudaErrors( hipCreateTextureObject( &_texture, &resDesc, &texDesc, NULL ));
}

ColorMap::~ColorMap()
{}

/** Deletes the cuda objects */
void ColorMap::clear()
{
    hipFreeArray( _array );
}

void ColorMap::upload( const lexis::render::ColorMap& colorMap )
{
    const auto& colors =
            colorMap.sampleColors< float >( 256, 0.0f, 256.0f, 0 );
    checkCudaErrors( hipMemcpyToArray( _array, 0, 0,
                                        colors.data(),
                                        colors.size() * sizeof(float4),
                                        hipMemcpyHostToDevice ));
}
}
}
